#include "hip/hip_runtime.h"

#include "graph_ray_backprojection.hpp"


// This flag activates timing of the code
#define DEBUG_TIME 0


#define EPSILON 0.000001

// Cuda error checking fucntion.
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        mexPrintf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort){
            hipDeviceReset();
            mexErrMsgIdAndTxt("MEX:graph_ray_projections", ".");
        }
    }
}


/**************************************************************************
 *********************** cross product in CUDA ****************************
 *************************************************************************/
__device__ __inline__ vec3d cross(const vec3d a,const vec3d b)
{
    vec3d c;
    c.x= a.y*b.z - a.z*b.y;
    c.y= a.z*b.x - a.x*b.z;
    c.z= a.x*b.y - a.y*b.x;
    return c;
}
/**************************************************************************
 *********************** Dot product in CUDA ******************************
 *************************************************************************/
__device__ __inline__ double dot(const vec3d a, const vec3d b)
{
    
    return a.x*b.x+a.y*b.y+a.z*b.z;
}


/**************************************************************************
 *********************** maximum value in a 4 valued array of floats*******
 *************************************************************************/
__device__ __inline__ float max4(float *t,int* indM){
    float max=0;
    *indM=-1;
    for(int i=0;i<4;i++){
        if (t[i]>max){
            max=t[i];
            *indM=i;
        }
    }
    return max;
}
/**************************************************************************
 ********* minimum nonzero value in a 4 valued array of float *************
 *************************************************************************/
__device__ __inline__ float min4nz(float *t){
    float min=1;
    for(int i=0;i<4;i++)
        min=(t[i]<min && t[i]!=0)?t[i]:min;
        return min;
}

/**************************************************************************
 ********* number of non zeroes in a 4 legth float array **** *************
 *************************************************************************/
__device__ __inline__ int nnz(float *t){
    int nz=0;
    for(int i=0;i<4;i++){
        if(t[i]>0){
            nz++;
        }
    }
    return nz;
    
}


/**************************************************************************
 *********************** Moller trumbore **********************************
 **************************************************************************/
__device__ __inline__ float moller_trumbore(const vec3 ray1, const vec3 ray2,
        const vec3d trip1,const vec3d trip2,const vec3d trip3, const float safetyEpsilon){
    

    
    
    vec3d direction,e1,e2;
    
    direction.x=ray2.x-ray1.x;     direction.y=ray2.y-ray1.y;     direction.z=ray2.z-ray1.z;
    e1.x       =trip2.x-trip1.x;   e1.y       =trip2.y-trip1.y;   e1.z       =trip2.z-trip1.z;
    e2.x       =trip3.x-trip1.x;   e2.y       =trip3.y-trip1.y;   e2.z       =trip3.z-trip1.z;
    
    
    vec3d q=cross(direction,e2);
    double a=dot(e1,q);
    if ((a>-EPSILON) & (a<EPSILON)){
        // the vector is parallel to the plane (the intersection is at infinity)
        return 0.0f;
    }
    
    double f=1/a;
    vec3d s;
    
    s.x=ray1.x-trip1.x;     s.y=ray1.y-trip1.y;     s.z=ray1.z-trip1.z;
    double u=f*dot(s,q);
    
    if (u<0.0-safetyEpsilon){
        // the intersection is outside of the triangle
        return 0.0f;
    }
    
    vec3d r=cross(s,e1);
    double v= f*dot(direction,r);
    
    if (v<0.0-safetyEpsilon || (u+v)>1.0+safetyEpsilon){
        // the intersection is outside of the triangle
        return 0.0;
    }
    return f*dot(e2,r);
    
    
    
}

/**************************************************************************
 ***************************Tetra-line intersection************************
 *************************************************************************/

__device__ __inline__ bool tetraLineIntersect(const unsigned long *elements,const float *vertices,
        const vec3 ray1, const vec3 ray2,
        const unsigned long elementId,float *t,bool computelenght,const float safetyEpsilon){
    
    unsigned long auxNodeId[4];
    auxNodeId[0]=elements[elementId*4+0];
    auxNodeId[1]=elements[elementId*4+1];
    auxNodeId[2]=elements[elementId*4+2];
    auxNodeId[3]=elements[elementId*4+3];
    
    
    vec3d triN1,triN2,triN3;
    
    float l1,l2,l3,l4;
    
    ///////////////////////////////////////////////////////////////////////
    // As modular arithmetic is bad on GPUs (flop-wise), I manually unroll the loop
    //for (int i=0;i<4;i++)
    ///////////////////////////////////////////////////////////////////////
    // Triangle
    triN1.x=vertices[auxNodeId[0]*3+0];    triN1.y=vertices[auxNodeId[0]*3+1];    triN1.z=vertices[auxNodeId[0]*3+2];
    triN2.x=vertices[auxNodeId[1]*3+0];    triN2.y=vertices[auxNodeId[1]*3+1];    triN2.z=vertices[auxNodeId[1]*3+2];
    triN3.x=vertices[auxNodeId[2]*3+0];    triN3.y=vertices[auxNodeId[2]*3+1];    triN3.z=vertices[auxNodeId[2]*3+2];
    //compute
    l1=moller_trumbore(ray1,ray2,triN1,triN2,triN3,safetyEpsilon);
    // Triangle
    triN1.x=vertices[auxNodeId[0]*3+0];    triN1.y=vertices[auxNodeId[0]*3+1];    triN1.z=vertices[auxNodeId[0]*3+2];
    triN2.x=vertices[auxNodeId[1]*3+0];    triN2.y=vertices[auxNodeId[1]*3+1];    triN2.z=vertices[auxNodeId[1]*3+2];
    triN3.x=vertices[auxNodeId[3]*3+0];    triN3.y=vertices[auxNodeId[3]*3+1];    triN3.z=vertices[auxNodeId[3]*3+2];
    //compute
    l2=moller_trumbore(ray1,ray2,triN1,triN2,triN3,safetyEpsilon);
    // Triangle
    triN1.x=vertices[auxNodeId[0]*3+0];    triN1.y=vertices[auxNodeId[0]*3+1];    triN1.z=vertices[auxNodeId[0]*3+2];
    triN2.x=vertices[auxNodeId[2]*3+0];    triN2.y=vertices[auxNodeId[2]*3+1];    triN2.z=vertices[auxNodeId[2]*3+2];
    triN3.x=vertices[auxNodeId[3]*3+0];    triN3.y=vertices[auxNodeId[3]*3+1];    triN3.z=vertices[auxNodeId[3]*3+2];
    //compute
    l3=moller_trumbore(ray1,ray2,triN1,triN2,triN3,safetyEpsilon);
    // Triangle
    triN1.x=vertices[auxNodeId[1]*3+0];    triN1.y=vertices[auxNodeId[1]*3+1];    triN1.z=vertices[auxNodeId[1]*3+2];
    triN2.x=vertices[auxNodeId[2]*3+0];    triN2.y=vertices[auxNodeId[2]*3+1];    triN2.z=vertices[auxNodeId[2]*3+2];
    triN3.x=vertices[auxNodeId[3]*3+0];    triN3.y=vertices[auxNodeId[3]*3+1];    triN3.z=vertices[auxNodeId[3]*3+2];
    //compute
    l4=moller_trumbore(ray1,ray2,triN1,triN2,triN3,safetyEpsilon);
    
    //dump

        //fuck branches, but what can I do ....
        if ((l1==0.0)&&(l2==0.0)&&(l3==0.0)&&(l4==0.0)){
            t[0]=0.0;t[1]=0.0;t[2]=0.0;t[3]=0.0;
            return false;
        }else{
            t[0]=l1;t[1]=l2;t[2]=l3;t[3]=l4;
            // find which one is the intersection
            return true;
        }
}

/**************************************************************************
 ***************************Intersection between line-box******************
 *************************************************************************/

__device__ bool rayBoxIntersect(const vec3 ray1, const vec3 ray2,const vec3 nodemin, const vec3 nodemax){
    vec3 direction;
    direction.x=ray2.x-ray1.x;
    direction.y=ray2.y-ray1.y;
    direction.z=ray2.z-ray1.z;
    
    float tmin,tymin,tzmin;
    float tmax,tymax,tzmax;
    if (direction.x >= 0){
        tmin = (nodemin.x - ray1.x) / direction.x;
        tmax = (nodemax.x - ray1.x) / direction.x;
        
    }else{
        tmin = (nodemax.x - ray1.x) / direction.x;
        tmax = (nodemin.x - ray1.x) / direction.x;
    }
    
    if (direction.y >= 0){
        tymin = (nodemin.y - ray1.y) / direction.y;
        tymax = (nodemax.y - ray1.y) / direction.y;
    }else{
        tymin = (nodemax.y - ray1.y) / direction.y;
        tymax = (nodemin.y - ray1.y) / direction.y;
    }
    
    if ( (tmin > tymax) || (tymin > tmax) ){
        return false;
    }
    
    if (tymin > tmin){
        tmin = tymin;
    }
    
    if (tymax < tmax){
        tmax = tymax;
    }
    
    if (direction.z >= 0){
        tzmin = (nodemin.z - ray1.z) / direction.z;
        tzmax = (nodemax.z - ray1.z) / direction.z;
    }else{
        tzmin = (nodemax.z - ray1.z) / direction.z;
        tzmax = (nodemin.z - ray1.z) / direction.z;
    }
    
    
    if ((tmin > tzmax) || (tzmin > tmax)){
        return false;
    }
    // If we wanted the ts as output
////
// if (tzmin > tmin){
//     tmin = tzmin;
// }
//
// if (tzmax < tmax){
//     tmax = tzmax;
// }
////
    return true;
}
/**************************************************************************
 ******Fucntion to detect the first triangle to expand the graph***********
 *************************************************************************/

__global__ void initXrays(const unsigned long* elements, const float* vertices,
        const unsigned long *boundary,const unsigned long nboundary,
        float * d_aux, Geometry geo,
        const vec3 source,const vec3 deltaU,const vec3 deltaV,const vec3 uvOrigin,const vec3 nodemin,const vec3 nodemax)
{
    
    
    unsigned long  y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long  x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long  idx =  x  * geo.nDetecV + y;
    if ((x>= geo.nDetecU) || (y>= geo.nDetecV))
        return;
    
    unsigned int pixelV =(unsigned int)geo.nDetecV- y-1;
    unsigned int pixelU =(unsigned int) x;

    
    // Compute detector position
    vec3 det;
    det.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    det.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    det.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    
    // Should we even try? if the ray does not cross the boundary, dont try
    bool crossBound=rayBoxIntersect(source, det, nodemin,nodemax);
    if (!crossBound){
        d_aux[idx]=-1.0f;
        return;
    }
    
    
    
    // Check intersection with all elements in the boudnary
    unsigned long notintersect=nboundary;
    float t[4];
    float t1,tinter=10000.0f;
    float safetyEpsilon=0.0000001f;
    long crossingID=-1;
    //Check with all elements, and keep the one that gives lowest parameter
    while(notintersect==nboundary){
        notintersect=0;
        
        for(unsigned long i=0 ;i<nboundary;i++){
            tetraLineIntersect(elements,vertices,source,det,boundary[i],t,true,safetyEpsilon);
            if (nnz(t)==0){
                notintersect++;
            }else{
                t1=min4nz(t);
                if (t1<tinter){
                    tinter=t1;
                    crossingID=i;
                }
            }
        }
        safetyEpsilon=safetyEpsilon*10;           
    } 

    d_aux[idx]=(float)crossingID;
    return;

}
/**************************************************************************
 ******************The mein projection fucntion ***************************
 *************************************************************************/

__global__ void graphBackproject(const unsigned long *elements, const float *vertices,const unsigned long *boundary,const long *neighbours, const float * d_proj, const float * d_auxInit, float * d_image, Geometry geo,
        vec3 source, vec3 deltaU, vec3 deltaV, vec3 uvOrigin){
    
    unsigned long  y = blockIdx.y * blockDim.y + threadIdx.y;
     //unsigned long  y = threadIdx.y * gridDim.y + blockIdx.y;

    unsigned long  x = blockIdx.x * blockDim.x + threadIdx.x;
     //unsigned long  x = threadIdx.x * gridDim.x + blockIdx.x;

    unsigned long  idx =  x  * geo.nDetecV + y;
    if ((x>= geo.nDetecU) || (y>= geo.nDetecV))
        return;
    
    
    unsigned int pixelV =(unsigned int)geo.nDetecV- y-1;
    unsigned int pixelU =(unsigned int) x;

    
    // Read initial position. Generate auxiliar variables for element tracking
    long current_element=(long)d_auxInit[idx];
    long previous_element;
    long aux_element;

    // for speed. Minimize reads
    float pixel_value=d_proj[idx];
    //  Get the coordinates of the detector for this kernel
    vec3 det;
    
    det.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    det.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    det.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    
    
    // If the current element is "none", then we are done, we are not itnersecting the mesh
    if (current_element==-1){
        //no need to do stuff
        return;
    }
    // initialize variables for the lengths and resutl
    float length,t1,t2;
    float t[4];
    int indM;
    bool isIntersect;
    
    
    // Lets compute the first intersection outside the main loop. 
    // The structure of this loop has to be identical to the one in InitXrays() or 
    // there is risk of not getting the same floating point value bit by bit. 
    float safeEpsilon=0.00001f;
    isIntersect=tetraLineIntersect(elements,vertices,source,det,boundary[current_element],t,true,0.0f);
    while(!isIntersect){
        isIntersect=tetraLineIntersect(elements,vertices,source,det,boundary[current_element],t,true,safeEpsilon);
        if (nnz(t)<=1){
            isIntersect=false;
            safeEpsilon*=10;
        }
    }
    // Reset the safety variable
    safeEpsilon=0.00001f;
    
    // Find the maximum and minimum non-zero intersection parameters
    t2=max4(t,&indM);
    t1=min4nz(t);

    // Lets get the ray (direction) and the current intersection length. 
    vec3 direction,p1,p2;
    direction.x=det.x-source.x;     direction.y=det.y-source.y;     direction.z=det.z-source.z;
    p2.x=direction.x* (t2);  p2.y=direction.y* (t2); p2.z=direction.z* (t2);
    p1.x=direction.x* (t1);  p1.y=direction.y* (t1); p1.z=direction.z* (t1);
    
    length=sqrt((p2.x-p1.x)*(p2.x-p1.x)+(p2.y-p1.y)*(p2.y-p1.y)+(p2.z-p1.z)*(p2.z-p1.z));
    
    
    // Start accumulating the result
    atomicAdd(&d_image[boundary[current_element]],length*pixel_value);
    // If t1 and t2 are the same, we need to make sure that the one we choose as 
    // t2 (the one that will lead us to the next element) is the correct one.
    // Otherwise we will go out of the image, and the code will end. 
    // This piece of code makes sure that is checked and swaps them otherwise.
    if(t1==t2){
        aux_element=neighbours[boundary[current_element]*4+indM];
        if(aux_element==-1){
            int auxind;
            for(int i=0;i<4;i++){
                if(indM!=i && t[i]==t1){
                    auxind=i;
                }
            }
            indM=auxind;
        }
    }
    
    // Grab the index of the next elements and save the current one for further checking
    previous_element=boundary[current_element];
    current_element=neighbours[boundary[current_element]*4+indM];
    // if its "none" then thats it, we are done.
    if (current_element==-1){
        return;
    }
    
    float sumt;
    unsigned long c=0;
    bool noNeighbours=false;
    while(!noNeighbours && c<5000){ // RANDOM safe distance, change to something sensible
        // c is a counter to avoid infinite loops
        c++;
        // Check intersections we now this one is intersected )because it shares a face with the previosu one that was intersected)
        isIntersect=tetraLineIntersect(elements,vertices,source,det,(unsigned int)current_element,t,true,0.0f);
        while(!isIntersect){
            // If intersection failed, then lets slightly increase the size of the triangle 
            // (not really, we increase the bounds of acceptable intersection values)
            // We can do it without safety becasue we already know it must happen.
            isIntersect=tetraLineIntersect(elements,vertices,source,det,(unsigned int)current_element,t,true,safeEpsilon);
            if (nnz(t)<=1){
                isIntersect=false;
                safeEpsilon*=10;
            }
        }
        safeEpsilon=0.00001f;
        
        // Find the maximum and minimum non-zero intersection parameters
        t2=max4(t,&indM);

        t1=min4nz(t);
// if they are very similar just treat them as if they were the same
// NOTE This was necesary in a previosu version, Its left here just in case its neeed again.        
//////
//         if (fabsf(t2-t1)<0.00000001){
//             t2=t1;
//             t[indM]=t1;
//         }
//////
        
        // Are they all zero?
        sumt=t[0]+t[1]+t[2]+t[3];
        if (sumt!=0.0){
           // compute intersection length and update result integral
            p2.x=direction.x* (t2);  p2.y=direction.y* (t2); p2.z=direction.z* (t2);
            p1.x=direction.x* (t1);  p1.y=direction.y* (t1); p1.z=direction.z* (t1);
            length=sqrt((p2.x-p1.x)*(p2.x-p1.x)+(p2.y-p1.y)*(p2.y-p1.y)+(p2.z-p1.z)*(p2.z-p1.z));
            atomicAdd(&d_image[current_element],length*pixel_value);
            // Now lets make sure we can find the next element correctly
            
            // If t1 and t2 are the same, we need to make sure that the one we choose as 
            // t2 (the one that will lead us to the next element) is the correct one.
            // Otherwise we will go backwards and get trapped in an infinite loop 
            // This piece of code makes sure this does not happen.
            if(t1==t2){
                
                aux_element=neighbours[current_element*4+indM];
                if(aux_element==previous_element){
                    int auxind;
                    for(int i=0;i<4;i++){
                        if(indM!=i && t[i]==t1){
                            auxind=i;
                        }
                    }
                    indM=auxind;
                }
            }
            // Update the elements
            previous_element=current_element;
            current_element=neighbours[current_element*4+indM];
            
            // if we are out then thats it, we are done.
            if (current_element==-1){
                
                return;
            }
            continue;
        }
        // If there was no intrsection, then we are out. Can this even happen?
        noNeighbours=true;
    }//endwhile
    
    // It should never get here, ever.
    return;
}
/**************************************************************************
 *********************** Main fucntion ************************************
 *************************************************************************/
void graphBackwardRay(float const * const  projections,  Geometry geo,
                    const double * angles,const unsigned int nangles,
                    const float* nodes,const unsigned long nnodes,
                    const unsigned long* elements,const unsigned long nelements,
                    const long* neighbours,const unsigned long nneighbours,
                    const unsigned long* boundary,const unsigned long nboundary,
                    float * result)
{
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    float time;
    float timecopy=0, timekernel=0,timeaux;
    hipEvent_t start, stop;
    
     if (DEBUG_TIME){
        
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }
    // First send all the relevant data to CUDA, and allocate enough memory for the result
    size_t num_bytes_img  = nelements*sizeof(float);
    
    float* d_image;
    hipMalloc((void **)&d_image,num_bytes_img);
    hipMemset(d_image,0,num_bytes_img);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    size_t num_bytes_proj = geo.nDetecU*geo.nDetecV* sizeof(float);
    float * d_proj;
    hipMalloc((void **)&d_proj,num_bytes_proj);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    float * d_auxInit;
    hipMalloc((void **)&d_auxInit,num_bytes_proj);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    size_t num_bytes_nodes = nnodes*3*sizeof(float);
    float * d_nodes;
    hipMalloc((void **)&d_nodes,num_bytes_nodes);
    hipMemcpy(d_nodes,nodes,num_bytes_nodes,hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    size_t num_bytes_elements = nelements*4*sizeof(unsigned long);
    unsigned long * d_elements;
    hipMalloc((void **)&d_elements,num_bytes_elements);
    hipMemcpy(d_elements,elements,num_bytes_elements,hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    size_t num_bytes_neighbours = nneighbours*4*sizeof(long);
    long * d_neighbours;
    hipMalloc((void **)&d_neighbours,num_bytes_neighbours);
    hipMemcpy(d_neighbours,neighbours,num_bytes_neighbours,hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    size_t num_bytes_boundary = nboundary*sizeof(unsigned long);
    unsigned long * d_boundary;
    hipMalloc((void **)&d_boundary,num_bytes_boundary);
    hipMemcpy(d_boundary,boundary,num_bytes_boundary,hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    if (DEBUG_TIME){
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        
        mexPrintf("Time to memcpy:  %3.1f ms \n", time);
    }
    // Replace by a reduction (?)
    vec3 nodemin, nodemax;
    nodemin.x=nodes[0];
    nodemin.y=nodes[1];
    nodemin.z=nodes[2];
    nodemax.x=nodes[0];
    nodemax.y=nodes[1];
    nodemax.z=nodes[2];
    
    for(unsigned long i=1;i<nnodes;i++){
        nodemin.x=(nodes[i*3+0]<nodemin.x)?nodes[i*3+0]:nodemin.x;
        nodemin.y=(nodes[i*3+1]<nodemin.y)?nodes[i*3+1]:nodemin.y;
        nodemin.z=(nodes[i*3+2]<nodemin.z)?nodes[i*3+2]:nodemin.z;
        nodemax.x=(nodes[i*3+0]>nodemax.x)?nodes[i*3+0]:nodemax.x;
        nodemax.y=(nodes[i*3+1]>nodemax.y)?nodes[i*3+1]:nodemax.y;
        nodemax.z=(nodes[i*3+2]>nodemax.z)?nodes[i*3+2]:nodemax.z;
    }
    
    // KERNEL TIME!
    int divU,divV;
    divU=8;
    divV=8;
    dim3 grid((geo.nDetecU+divU-1)/divU,(geo.nDetecV+divV-1)/divV,1);
    dim3 block(divU,divV,1);
    
    vec3  deltaU, deltaV, uvOrigin;
    vec3 source;
    for (unsigned int i=0;i<nangles;i++){
        if (DEBUG_TIME){
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
        }

        gpuErrchk(hipMemcpyAsync(d_proj,&projections[geo.nDetecU*geo.nDetecV*i],num_bytes_proj,hipMemcpyHostToDevice));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        geo.alpha=angles[i*3];
        geo.theta=angles[i*3+1];
        geo.psi  =angles[i*3+2];
        computeGeomtricParams(geo, &source,&deltaU, &deltaV,&uvOrigin,i);
        if (DEBUG_TIME){
            
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&timeaux, start, stop);
            timecopy+=timeaux;
            
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
        }
        initXrays << <grid,block >> >(d_elements,d_nodes,d_boundary,nboundary,d_auxInit, geo, source,deltaU, deltaV,uvOrigin,nodemin,nodemax);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        graphBackproject<< <grid,block >> >(d_elements,d_nodes,d_boundary,d_neighbours,d_proj,d_auxInit,d_image, geo,source,deltaU,deltaV,uvOrigin);
        
        gpuErrchk(hipPeekAtLastError()); 
        gpuErrchk(hipDeviceSynchronize());
        
 
        if (DEBUG_TIME){
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&timeaux, start, stop);
            timekernel+=timeaux;
        }
    }
    
    
    if (DEBUG_TIME){
        mexPrintf("Time of Kenrel:  %3.1f ms \n", timekernel);
        mexPrintf("Time of memcpy to Host:  %3.1f ms \n", timecopy);
        
    }
    
    gpuErrchk(hipMemcpy(result, d_image, num_bytes_img, hipMemcpyDeviceToHost));

    if (DEBUG_TIME){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    }
//     cudaGraphFree(&tempHostGraph,&tempHostElement,&tempHostNode);
    hipFree(d_proj);
    hipFree(d_auxInit);
    hipFree(d_image);
    hipFree(d_nodes);
    hipFree(d_neighbours);
    hipFree(d_elements);
    hipFree(d_boundary);
    if (DEBUG_TIME){
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        
        mexPrintf("Time to free:  %3.1f ms \n", time);
    }
    return;
    
    
}



// TODO: quite a lot of geometric transforms.
void computeGeomtricParams(const Geometry geo,vec3 * source, vec3* deltaU, vec3* deltaV, vec3* originUV,unsigned int idxAngle){
    
    vec3 auxOriginUV;
    vec3 auxDeltaU;
    vec3 auxDeltaV;
    auxOriginUV.x=-(geo.DSD[idxAngle]-geo.DSO[idxAngle]);
    // top left
    auxOriginUV.y=-geo.sDetecU/2+/*half a pixel*/geo.dDetecU/2;
    auxOriginUV.z=geo.sDetecV/2-/*half a pixel*/geo.dDetecV/2;
    
    //Offset of the detector
    auxOriginUV.y=auxOriginUV.y+geo.offDetecU[idxAngle];
    auxOriginUV.z=auxOriginUV.z+geo.offDetecV[idxAngle];
    
    // Change in U
    auxDeltaU.x=auxOriginUV.x;
    auxDeltaU.y=auxOriginUV.y+geo.dDetecU;
    auxDeltaU.z=auxOriginUV.z;
    //Change in V
    auxDeltaV.x=auxOriginUV.x;
    auxDeltaV.y=auxOriginUV.y;
    auxDeltaV.z=auxOriginUV.z-geo.dDetecV;
    
    vec3 auxSource;
    auxSource.x=geo.DSO[idxAngle];
    auxSource.y=0;
    auxSource.z=0;
    
    // rotate around axis.
    eulerZYZ(geo,&auxOriginUV);
    eulerZYZ(geo,&auxDeltaU);
    eulerZYZ(geo,&auxDeltaV);
    eulerZYZ(geo,&auxSource);
    
    // Offset image (instead of offseting image, -offset everything else)
    auxOriginUV.x  =auxOriginUV.x-geo.offOrigX[idxAngle];     auxOriginUV.y  =auxOriginUV.y-geo.offOrigY[idxAngle];     auxOriginUV.z  =auxOriginUV.z-geo.offOrigZ[idxAngle];
    auxDeltaU.x=auxDeltaU.x-geo.offOrigX[idxAngle];           auxDeltaU.y=auxDeltaU.y-geo.offOrigY[idxAngle];           auxDeltaU.z=auxDeltaU.z-geo.offOrigZ[idxAngle];
    auxDeltaV.x=auxDeltaV.x-geo.offOrigX[idxAngle];           auxDeltaV.y=auxDeltaV.y-geo.offOrigY[idxAngle];           auxDeltaV.z=auxDeltaV.z-geo.offOrigZ[idxAngle];
    auxSource.x=auxSource.x-geo.offOrigX[idxAngle];           auxSource.y=auxSource.y-geo.offOrigY[idxAngle];           auxSource.z=auxSource.z-geo.offOrigZ[idxAngle];
    
    auxDeltaU.x=auxDeltaU.x-auxOriginUV.x;  auxDeltaU.y=auxDeltaU.y-auxOriginUV.y; auxDeltaU.z=auxDeltaU.z-auxOriginUV.z;
    auxDeltaV.x=auxDeltaV.x-auxOriginUV.x;  auxDeltaV.y=auxDeltaV.y-auxOriginUV.y; auxDeltaV.z=auxDeltaV.z-auxOriginUV.z;
    
    *originUV=auxOriginUV;
    *deltaU=auxDeltaU;
    *deltaV=auxDeltaV;
    *source=auxSource;
    
    return;
}

void eulerZYZ(Geometry geo,  vec3* point){
    vec3 auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=(+cos(geo.alpha)*cos(geo.theta)*cos(geo.psi)-sin(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-cos(geo.alpha)*cos(geo.theta)*sin(geo.psi)-sin(geo.alpha)*cos(geo.psi))*auxPoint.y+
            cos(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->y=(+sin(geo.alpha)*cos(geo.theta)*cos(geo.psi)+cos(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-sin(geo.alpha)*cos(geo.theta)*sin(geo.psi)+cos(geo.alpha)*cos(geo.psi))*auxPoint.y+
            sin(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->z=-sin(geo.theta)*cos(geo.psi)*auxPoint.x+
            sin(geo.theta)*sin(geo.psi)*auxPoint.y+
            cos(geo.theta)*auxPoint.z;
    
    
    
    
}